#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <netinet/in.h>
extern "C" {
#include "firewall_kernels.h"
}
#include "rules.h"

__global__ void process_pkt1(char* input_buf,
                             char* output_buf,
                             int* len,
                             int num_pkts,
                             int buf_len) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;

    if (tx < num_pkts) {
        int pkt_start = len[tx];
        int pkt_end = len[tx + 1];
        int pkt_len = pkt_end - pkt_start;

        // Copy the packet data from input_buf to output_buf
        for (int i = 0; i < pkt_len; i++) {
            output_buf[pkt_start + i] = input_buf[pkt_start + i];
        }
        // output_buf = input_buf;
    }
}

void run_firewall(char* input_buf,
                  char* output_buf,
                  int* len,
                  int num_pkts,
                  int buf_len,
                  unsigned long* rules,
                  int num_lines,
                  unsigned int* nat_table,
                  unsigned long* nat_set) {


    // Allocate device memory
    char* d_input_buf;
    char* d_output_buf;
    int* d_len;

    hipMalloc((void**)&d_input_buf, buf_len * sizeof(char));
    hipMalloc((void**)&d_output_buf, buf_len * sizeof(char));
    hipMalloc((void**)&d_len, (num_pkts + 1) * sizeof(int));

    // Copy input buffer and len array to device memory
    hipMemcpy(d_input_buf, input_buf, buf_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_len, len, (num_pkts + 1) * sizeof(int), hipMemcpyHostToDevice);

    // Launch process_pkts kernel with appropriate block and thread configuration
    int threads_per_block = 64;  // You can adjust this as needed
    int num_blocks = (num_pkts + threads_per_block - 1) / threads_per_block;
    process_pkt1<<<num_blocks, threads_per_block>>>(d_input_buf, d_output_buf, d_len, num_pkts, buf_len);
    hipDeviceSynchronize();

    // Copy output buffer to host memory
    hipMemcpy(output_buf, d_output_buf, buf_len * sizeof(char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input_buf);
    hipFree(d_output_buf);
    hipFree(d_len);
}


#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <netinet/in.h>
extern "C" {
#include "firewall_kernels.h"
}
#include "rules.h"

// Static pointers for device memory
static char* d_input_buf = nullptr;
static char* d_output_buf = nullptr;
static int* d_len = nullptr;

__global__ void process_pkt1(char** input_buf,
                             char** output_buf,
                             int* len,
                             int num_pkts,
                             int buf_len) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;

    if (tx < num_pkts) {
        int pkt_start = len[tx];
        int pkt_end = len[tx + 1];
        int pkt_len = pkt_end - pkt_start;

        // Copy the packet data from input_buf to output_buf
        for (int i = 0; i < pkt_len; i++) {
            output_buf[pkt_start + i] = input_buf[pkt_start + i];
        }
    }
}

void run_firewall(char** input_buf,
                  char** output_buf,
                  int* len,
                  int num_pkts,
                  int buf_len,
                  unsigned long* rules,
                  int num_lines,
                  unsigned int* nat_table,
                  unsigned long* nat_set) {

    
    // Launch process_pkts kernel with appropriate block and thread configuration
    int threads_per_block = 64;  // You can adjust this as needed
    int num_blocks = (num_pkts + threads_per_block - 1) / threads_per_block;
    process_pkt1<<<num_blocks, threads_per_block>>>((char **)input_buf, (char **)output_buf, d_len, num_pkts, buf_len);
    hipDeviceSynchronize();
}
