#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <arpa/inet.h>

extern "C" {
#include "nat.h"
}

int allocate_nat_table(unsigned int **d_nat) { 
    unsigned int *d_nat_tmp;
    hipMalloc((void**)&d_nat_tmp, 2 * 10 * 10000 * sizeof(unsigned int));
    hipMemset(d_nat_tmp, 0, 2 * 10 * 10000 * sizeof(unsigned int));

    *d_nat = d_nat_tmp;

    return 0;
}

int allocate_nat_set(unsigned long **d_nat) { 
    unsigned long *d_nat_tmp;
    hipMalloc((void**)&d_nat_tmp, 10 * 10000 * sizeof(unsigned long));
    hipMemset(d_nat_tmp, 0, 10 * 10000 * sizeof(unsigned long));

    *d_nat = d_nat_tmp;

    return 0;
}

int free_nat(unsigned int *d_nat_table, unsigned long *d_nat_set) {
    hipFree(d_nat_table);    
    hipFree(d_nat_set);
    return 0;
}
